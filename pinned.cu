// memory page is managed by OS
// "unpageable" means cuda directly accesses physical RAM
// to access pageable memory allocated by like malloc()
//   pageable memory -> pinned memory -> device memory
// if we directly operate on pinned memory
//   then we save transfer from pageable memory to pinned memory

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
void HandleError(hipError_t err, const char *file, int line )
{
  if (err != hipSuccess)
  {
    printf( "%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit( EXIT_FAILURE );
  }
}

int main()
{
  unsigned int N = 256 * 1024 * 1024;

  float *d = NULL;
  HANDLE_ERROR( hipMalloc((void**)&d, N * sizeof(float)) );
  
  hipEvent_t startEvent, stopEvent;
  HANDLE_ERROR( hipEventCreate(&startEvent) );
  HANDLE_ERROR( hipEventCreate(&stopEvent) );

  float *h_a = NULL, *h_b = NULL;
  float time = 0;

  // test on pageable memory
  h_a = (float *) malloc(N * sizeof(float));
  h_b = (float *) malloc(N * sizeof(float));

  HANDLE_ERROR( hipEventRecord(startEvent, 0) );
  HANDLE_ERROR( hipMemcpy(d, h_a, N * sizeof(float), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipEventRecord(stopEvent, 0) );
  HANDLE_ERROR( hipEventSynchronize(stopEvent) );
  HANDLE_ERROR( hipEventElapsedTime(&time, startEvent, stopEvent) );
  printf("Pageable, host to device, %.3fGB/s\n", 
          N * sizeof(float) * 1000.0 / (1024 * 1024 * 1024 * time));

  HANDLE_ERROR( hipEventRecord(startEvent, 0) );
  HANDLE_ERROR( hipMemcpy(h_b, d, N * sizeof(float), hipMemcpyDeviceToHost) );
  HANDLE_ERROR( hipEventRecord(stopEvent, 0) );
  HANDLE_ERROR( hipEventSynchronize(stopEvent) );
  HANDLE_ERROR( hipEventElapsedTime(&time, startEvent, stopEvent) );
  printf("Pageable, device to host, %.3fGB/s\n", 
          N * sizeof(float) * 1000.0 / (1024 * 1024 * 1024 * time));

  free(h_a);
  free(h_b);
  h_a = NULL; 
  h_b = NULL;

  // test on pinned memory
  HANDLE_ERROR( hipHostMalloc((void**)&h_a, N * sizeof(float), hipHostMallocDefault) );
  HANDLE_ERROR( hipHostMalloc((void**)&h_b, N * sizeof(float), hipHostMallocDefault) );

  HANDLE_ERROR( hipEventRecord(startEvent, 0) );
  HANDLE_ERROR( hipMemcpy(d, h_a, N * sizeof(float), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipEventRecord(stopEvent, 0) );
  HANDLE_ERROR( hipEventSynchronize(stopEvent) );
  HANDLE_ERROR( hipEventElapsedTime(&time, startEvent, stopEvent) );
  printf("Pinned, host to device, %.3fGB/s\n", 
          N * sizeof(float) * 1000.0 / (1024 * 1024 * 1024 * time));

  HANDLE_ERROR( hipEventRecord(startEvent, 0) );
  HANDLE_ERROR( hipMemcpy(h_b, d, N * sizeof(float), hipMemcpyDeviceToHost) );
  HANDLE_ERROR( hipEventRecord(stopEvent, 0) );
  HANDLE_ERROR( hipEventSynchronize(stopEvent) );
  HANDLE_ERROR( hipEventElapsedTime(&time, startEvent, stopEvent) );
  printf("Pinned, device to host, %.3fGB/s\n", 
          N * sizeof(float) * 1000.0 / (1024 * 1024 * 1024 * time));

  hipHostFree(h_a);
  hipHostFree(h_b);


  // clean up
  hipFree(d);
  HANDLE_ERROR( hipEventDestroy(startEvent) );
  HANDLE_ERROR( hipEventDestroy(stopEvent) );  
  return 0;
}
